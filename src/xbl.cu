#include "hip/hip_runtime.h"
/*
 * Fermat
 *
 * Copyright (c) 2016-2019, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "xbl.h"

#include <cugar/basic/cuda/arch.h>
#include <cugar/linalg/vector.h>
#include <cugar/spherical/mappings.h>

namespace {

FERMAT_HOST_DEVICE
float norm_diff(const cugar::Vector3f a, const cugar::Vector3f b)
{
	const float d = cugar::max(1e-8f, cugar::dot(a, b));
	return 1.0f - d;
}

__device__ float myexp(float in)
{
	float tmp = 0.0f;
	if (fabsf(in) < 1.0f)
	{
		tmp = (1.0f + 0.45f*in);
	}
	return tmp * tmp;
//	return __expf(in);
}

} // anonymous

/// perform a step of cross-bilateral filtering
///
__global__
void XBL_mad_kernel(
		  FBufferChannelView	dst,
	const uint32				op,
	const FBufferChannelView	w_img,
	const float					w_min,
	const FBufferChannelView	img,
	const GBufferView			gb,
	const float*				var,
	const XBLParams				params,
	const uint32				filter_radius,
	const uint32				step_size,
	const TiledSequenceView		sequence)
{
	const int32 x = threadIdx.x + blockIdx.x * blockDim.x;
	const int32 y = threadIdx.y + blockIdx.y * blockDim.y;

	// check whether this pixel is in range
	if (x >= dst.res_x ||
		y >= dst.res_y)
		return;

	const cugar::Vector4f  weightCenter		= cugar::max( cugar::Vector4f(w_img(x, y)), w_min );
	const cugar::Vector4f  imgCenter		= img(x, y);

	const cugar::Vector4f  colorCenter		=
		(op & kFilterOpModulateInput)   ? imgCenter * weightCenter :
		(op & kFilterOpDemodulateInput) ? imgCenter / weightCenter :
		imgCenter;

	const cugar::Vector4f  packed_geo		= gb.geo(x, y);
	const cugar::Vector3f  normalCenter		= GBufferView::unpack_normal(packed_geo);
	const cugar::Vector3f  positionCenter	= GBufferView::unpack_pos(packed_geo);

	// check whether this pixel represents a miss (TODO: for stochastic effects, we might want to do some filtering in this case too...)
	if (GBufferView::is_miss(packed_geo))
	{
		cugar::Vector4f r  = (op & kFilterOpAddMode) ? dst(x, y) : cugar::Vector4f(0.0f);

		r +=
			(op & kFilterOpModulateOutput)	 ?	colorCenter * weightCenter :
			(op & kFilterOpDemodulateOutput) ?	colorCenter / weightCenter :
												colorCenter;

		dst(x, y) = r;
		return;
	}

	const float posRadius = 20 * cugar::min(
		cugar::length(params.U) / img.res_x,
		cugar::length(params.V) / img.res_y) * dot( positionCenter - params.E, params.W ) / cugar::square_length(params.W);

	const float variance	= var ? var[x + y * img.res_x] : 1.0f;
	const float phiNormal	= params.phi_normal * step_size * step_size;
	const float phiPosition	= params.phi_position / (posRadius*posRadius);
	const float phiColor	= params.phi_color / cugar::max( 1.0e-3f, cugar::sqr(variance) );
	const float nThreshold  = 0.9f;

	float			sumWeight = 0.0;
	cugar::Vector3f	sumColor = cugar::Vector3f(0.0f);

    const float sigma  = 10.0f;
    const float sigma2 = sigma*sigma;

	//for (int yy = -int(filter_radius); yy <= int(filter_radius); yy++)
	for (uint32 s = 0; s < params.taps; ++s)
	{
		const float u = cugar::randfloat(s,0) + sequence.shift(x, y, 0u);
		const float v = cugar::randfloat(s,1) + sequence.shift(x, y, 1u);

		const cugar::Vector2f xy =
			s == 0	? cugar::Vector2f(0.0f)								// make sure to sample the central pixel
					: cugar::square_to_unit_disk(cugar::Vector2f(u,v));
		const float xx = xy.x * filter_radius;
		const float yy = xy.y * filter_radius;

		//for (int xx = -int(filter_radius); xx <= int(filter_radius); xx++)
		{
			const int2 p = make_int2(x + xx * step_size, y + yy * step_size);
			const bool inside =
				/*__all*/(p.x >= 0 && p.y >= 0) &&
				/*__all*/(p.x < img.res_x && p.y < img.res_y);

			if (inside)
			{
				const float d2 = (xx*xx + yy*yy) / sigma2;

				const cugar::Vector4f  weightP = cugar::max( cugar::Vector4f(w_img(p)), w_min );
				const cugar::Vector4f  imgP	   = img(p);

				const cugar::Vector4f  colorP =
					(op & kFilterOpModulateInput)	? imgP * weightP :
					(op & kFilterOpDemodulateInput)	? imgP / weightP :
					imgP;

				const cugar::Vector4f  geoP = gb.geo(p);
				const cugar::Vector3f  normalP = GBufferView::unpack_normal(geoP);
				const cugar::Vector3f  positionP = GBufferView::unpack_pos(geoP);

				if (GBufferView::is_miss(geoP) == false)
				{
					// check whether we have to skip this pixel
					if (dot(normalP, normalCenter) < nThreshold)
						continue;

					// compute the normal weight
					const float		wNormal			= norm_diff(normalP, normalCenter) * phiNormal;

					// compute the color weight
					cugar::Vector3f diffCol			= colorP.xyz() - colorCenter.xyz();
					const float		wColor			= cugar::dot(diffCol, diffCol) * phiColor;

					// compute the positional weight
					cugar::Vector3f	diffPosition	= (positionP - positionCenter);
					const float		wPosition		= dot(diffPosition, diffPosition) * phiPosition;

					const float w = myexp(0.0
						- d2
						- cugar::max(wPosition, 0.0f)
						- cugar::max(wNormal, 0.0f)
						- cugar::max(wColor, 0.0f)
					);

					sumWeight += w;
					sumColor  += w * colorP.xyz();
				}
			}
		}
	}

	cugar::Vector4f r  = (op & kFilterOpAddMode) ? dst(x, y) : cugar::Vector4f(0.0f);

	cugar::Vector4f c = (sumWeight ? cugar::Vector4f(sumColor / sumWeight, colorCenter.w) : colorCenter);

	r +=
		(op & kFilterOpModulateOutput)		? c * weightCenter :
		(op & kFilterOpDemodulateOutput)	? c / weightCenter :
		c;

	dst(x, y) = r;
}

// perform a step of cross-bilateral filtering, multiplying the result by a weight and adding it to the output, i.e. solving:
//
//   dst += w_img * xbl(img)
//
void XBL(
	FBufferChannelView			dst,
	const FilterOp				op,
	const FBufferChannelView	w_img,
	const float					w_min,
	const FBufferChannelView	img,
	const GBufferView			gb,
	const float*				var,
	const XBLParams				params,
	const uint32				filter_radius,
	const uint32				step_size,
	const TiledSequenceView		sequence)
{
	dim3 blockSize(32, 4);
	dim3 gridSize(cugar::divide_ri(dst.res_x, blockSize.x), cugar::divide_ri(dst.res_y, blockSize.y));

	XBL_mad_kernel<< < gridSize, blockSize >> > (dst, op, w_img, w_min, img, gb, var, params, filter_radius, step_size, sequence);
	CUDA_CHECK(cugar::cuda::sync_and_check_error("XBL_mad_kernel"));
}
