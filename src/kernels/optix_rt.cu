#include "hip/hip_runtime.h"
/*
 * Fermat
 *
 * Copyright (c) 2018-2019, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "optix_payload.h"
#include "optix_common_variables.h"
#include "optix_attributes.h"

#include <ray.h>

//-----------------------------------------------------------------------------
// Ray generation program: plain ray tracing kernel, without masking
//-----------------------------------------------------------------------------

rtBuffer<float4>	g_ray_buffer;
rtBuffer<float4>	g_hit_buffer;

RT_PROGRAM
void program_tmin_ray_generation()
{
	const uint32 idx = g_launch_index;

	const float4 f1 = g_ray_buffer[idx*2 + 0];
	const float4 f2 = g_ray_buffer[idx*2 + 1];
	
	Ray ray;
	ray.origin.x = f1.x;
	ray.origin.y = f1.y;
	ray.origin.z = f1.z;
	ray.tmin     = f1.w;

	ray.dir.x = f2.x;
	ray.dir.y = f2.y;
	ray.dir.z = f2.z;
	ray.tmax  = f2.w;

	// trace the ray generated at the previous bounce
	Payload payload(
		-1.0f,	// t,
		-1,		// triangle id
		0.0f,	// u
		0.0f,	// v
		0x0u );	// mask

	rtTrace( g_top_object, optix::make_Ray(ray.origin, ray.dir, 2u /* ray type 2: no masking */, ray.tmin, ray.tmax), payload );

	const Hit hit = Hit(payload);

	g_hit_buffer[idx] = make_float4(
		hit.t,
		__uint_as_float(hit.triId),
		hit.u,
		hit.v
	);
}


//-----------------------------------------------------------------------------
// Ray generation program: plain ray tracing kernel, with masking
//-----------------------------------------------------------------------------

RT_PROGRAM
void program_masked_ray_generation()
{
	const uint32 idx = g_launch_index;

	const float4 f1 = g_ray_buffer[idx*2 + 0];
	const float4 f2 = g_ray_buffer[idx*2 + 1];
	
	MaskedRay ray;
	ray.origin.x = f1.x;
	ray.origin.y = f1.y;
	ray.origin.z = f1.z;
	ray.mask     = __float_as_uint(f1.w);

	ray.dir.x = f2.x;
	ray.dir.y = f2.y;
	ray.dir.z = f2.z;
	ray.tmax  = f2.w;

	// trace the ray generated at the previous bounce
	Payload payload(
		-1.0f,	// t,
		-1,		// triangle id
		0.0f,	// u
		0.0f,	// v
		ray.mask );	// mask

	rtTrace( g_top_object, optix::make_Ray(ray.origin, ray.dir, 0u /* ray type 0: masking */, 0.0f, ray.tmax), payload );

	const Hit hit = Hit(payload);

	g_hit_buffer[idx] = make_float4(
		hit.t,
		__uint_as_float(hit.triId),
		hit.u,
		hit.v
	);
}

//-----------------------------------------------------------------------------
// Ray generation program: shadow ray tracing kernel, with masking, reporting
// the first valid hit (not necessarily the closest)
//-----------------------------------------------------------------------------

RT_PROGRAM
void program_masked_shadow_ray_generation()
{
	const uint32 idx = g_launch_index;

	const float4 f1 = g_ray_buffer[idx*2 + 0];
	const float4 f2 = g_ray_buffer[idx*2 + 1];
	
	MaskedRay ray;
	ray.origin.x = f1.x;
	ray.origin.y = f1.y;
	ray.origin.z = f1.z;
	ray.mask     = __float_as_uint(f1.w);

	ray.dir.x = f2.x;
	ray.dir.y = f2.y;
	ray.dir.z = f2.z;
	ray.tmax  = f2.w;

	// trace the ray generated at the previous bounce
	ShadowPayload hit( ray.mask, false );

	rtTrace( g_top_object, optix::make_Ray(ray.origin, ray.dir, 1u /* ray type 1: shadow w/ masking */, 0.0f, ray.tmax), hit );

	// TODO / FIXME: for now all of the fields are invalid, as they just fake a valid intersection
	g_hit_buffer[idx] = make_float4(
		hit ? 1.0f : -1.0f,
		hit ? __uint_as_float(1u) : __uint_as_float(-1u),
		0.0f,
		0.0f
	);
}

//-----------------------------------------------------------------------------
// Ray generation program: shadow ray tracing kernel, with masking, reporting
// just a binary hit / no hit bit
//-----------------------------------------------------------------------------

rtDeclareVariable(uint32*, g_binary_hits, ,   );

RT_PROGRAM
void program_masked_shadow_binary_ray_generation()
{
	const uint32 idx = g_launch_index;

	const float4 f1 = g_ray_buffer[idx*2 + 0];
	const float4 f2 = g_ray_buffer[idx*2 + 1];
	
	MaskedRay ray;
	ray.origin.x = f1.x;
	ray.origin.y = f1.y;
	ray.origin.z = f1.z;
	ray.mask     = __float_as_uint(f1.w);

	ray.dir.x = f2.x;
	ray.dir.y = f2.y;
	ray.dir.z = f2.z;
	ray.tmax  = f2.w;

	// trace the ray generated at the previous bounce
	ShadowPayload hit( ray.mask, false );

	rtTrace( g_top_object, optix::make_Ray(ray.origin, ray.dir, 1u /* ray type 1: shadow w/ masking */, 0.0f, ray.tmax), hit );

	const uint32 word_idx = idx >> 5;
	const uint32 word_bit = idx & 31u;

	const uint32 word_mask = 1u << word_bit;

	if (hit) atomicOr(  g_binary_hits + word_idx,  word_mask );
	else     atomicAnd( g_binary_hits + word_idx, ~word_mask );
}
