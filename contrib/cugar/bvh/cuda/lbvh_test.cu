#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011, NVIDIA Corporation
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above copyright
 *     notice, this list of conditions and the following disclaimer in the
 *     documentation and/or other materials provided with the distribution.
 *   * Neither the name of NVIDIA Corporation nor the
 *     names of its contributors may be used to endorse or promote products
 *     derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <sandbox/bvh/cuda/lbvh_builder.h>
#include <sandbox/radixtree/radixtree.h>
#include <sandbox/sampling/random.h>
#include <sandbox/basic/timer.h>
#include <sandbox/bintree/bintree_visitor.h>
#include <sandbox/tree/cuda/reduce.h>

namespace sandbox {

struct bbox_functor
{
    SANDBOX_HOST_DEVICE Bbox4f operator() (
        const Vector4f op1,
        const Vector4f op2) const
    {
        Bbox4f result;
        result.insert( op1 );
        result.insert( op2 );
        return result;
    }
    SANDBOX_HOST_DEVICE Bbox4f operator() (
        const Bbox4f op1,
        const Bbox4f op2) const
    {
        Bbox4f result;
        result.insert( op1 );
        result.insert( op2 );
        return result;
    }
};

void lbvh_test()
{
    fprintf(stderr, "lbvh test... started\n");

    const uint32 n_points = 4*1024*1024;
    const uint32 n_tests = 100;

    thrust::host_vector<Vector4f> h_points( n_points );

    Random random;
    for (uint32 i = 0; i < n_points; ++i)
        h_points[i] = Vector4f( random.next(), random.next(), random.next(), 1.0f );

    thrust::device_vector<Vector4f> d_points( h_points );
    thrust::device_vector<Vector4f> d_unsorted_points( h_points );

    sandbox::vector<device_tag,Bvh_node> bvh_nodes;
    sandbox::vector<device_tag,uint32>   bvh_index;
    sandbox::vector<device_tag,uint32>   bvh_skip_nodes;
    sandbox::vector<device_tag,uint32>   bvh_parents;

    cuda::LBVH_builder<uint64> builder( &bvh_nodes, &bvh_index );
    builder.set_parents( &bvh_parents );
    builder.set_skip_nodes( &bvh_skip_nodes );

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    float time = 0.0f;

    for (uint32 i = 0; i <= n_tests; ++i)
    {
        d_points = d_unsorted_points;
        hipDeviceSynchronize();

        float dtime;
        hipEventRecord( start, 0 );

        builder.build(
            Bbox3f( Vector3f(0.0f), Vector3f(1.0f) ),
            d_points.begin(),
            d_points.end(),
            16u );

        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &dtime, start, stop );

        if (i) // skip the first run
            time += dtime;
    }
    time /= 1000.0f * float(n_tests);

    hipEventDestroy( start );
    hipEventDestroy( stop );

    {
        sandbox::vector<host_tag,uint64>   h_codes( builder.m_codes );
        sandbox::vector<host_tag,Bvh_node> h_nodes;
        sandbox::vector<host_tag,uint32>   h_skip_nodes;

        const uint32 n_codes = n_points;

        Bintree_writer<Bvh_node,host_tag> tree_writer;
        tree_writer.set_nodes( &h_nodes );
        tree_writer.set_skip_nodes( &h_skip_nodes );
        generate_radix_tree(
            n_codes,
            &h_codes[0],
            60,
            16u,
            false,
            tree_writer );

        sandbox::vector<host_tag,Bvh_node> d_nodes      = bvh_nodes;
        sandbox::vector<host_tag,uint32>   d_skip_nodes = bvh_skip_nodes;

        // traverse both trees top-down to see whether there's any inconsistencies...
        uint32 h_node_id = 0;
        uint32 d_node_id = 0;
        uint32 node_index = 0;
        uint32 leaf_index = 0;

        while (h_node_id != uint32(-1))
        {
            if (d_node_id == uint32(-1))
            {
                fprintf(stderr, "device node is invalid!\n");
                break;
            }

            Bvh_node h_node = h_nodes[ h_node_id ];
            Bvh_node d_node = d_nodes[ d_node_id ];

            if (h_node.is_leaf() != d_node.is_leaf())
            {
                fprintf(stderr, "host node and device node have different topology! (%u) (%s, %s)\n", node_index, h_node.is_leaf() ? "leaf" : "split", d_node.is_leaf() ? "leaf" : "split" );
                break;
            }

            if (h_node.is_leaf())
            {
                const uint2 h_leaf = h_node.get_leaf_range() ;
                const uint2 d_leaf = d_node.get_leaf_range() ;

                if (h_leaf.x != d_leaf.x ||
                    h_leaf.y != d_leaf.y)
                {
                    fprintf(stderr, "host and device leaves differ! [%u,%u) != [%u,%u) (%u:%u)\n",
                        h_leaf.x, h_leaf.y,
                        d_leaf.x, d_leaf.y,
                        node_index, leaf_index );
                    break;
                }

                h_node_id = h_skip_nodes[ h_node_id ];
                d_node_id = d_skip_nodes[ d_node_id ];

                leaf_index++;
            }
            else
            {
                h_node_id = h_node.get_child(0);
                d_node_id = d_node.get_child(0);
            }

            node_index++;
        }
   }

    fprintf(stderr, "lbvh test... done\n");
    fprintf(stderr, "  time       : %f ms\n", time * 1000.0f );
    fprintf(stderr, "  points/sec : %f M\n", (n_points / time) / 1.0e6f );

    fprintf(stderr, "  nodes  : %u\n", builder.m_node_count );
    fprintf(stderr, "  leaves : %u\n", builder.m_leaf_count );
    for (uint32 level = 0; level < 60; ++level)
        fprintf(stderr, "  level %u : %u nodes\n", level, builder.m_levels[level+1] - builder.m_levels[level] );

    fprintf(stderr, "lbvh bbox reduction test... started\n");

    Bintree_visitor<Bvh_node> bvh;
    bvh.set_node_count( builder.m_node_count );
    bvh.set_leaf_count( builder.m_leaf_count );
    bvh.set_nodes( raw_pointer( bvh_nodes ) );
    bvh.set_parents( raw_pointer( bvh_parents ) );

    sandbox::vector<device_tag,Bbox4f> bvh_bboxes( builder.m_node_count );

    hipEventCreate( &start );
    hipEventCreate( &stop );

    time = 0.0f;

    for (uint32 i = 0; i <= n_tests; ++i)
    {
        float dtime;
        hipEventRecord( start, 0 );

        cuda::tree_reduce(
            bvh,
            raw_pointer( d_points ),
            (Bbox4f* volatile)raw_pointer( bvh_bboxes ),
            bbox_functor(),
            Bbox4f() );

        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &dtime, start, stop );

        if (i) // skip the first run
            time += dtime;
    }
    time /= 1000.0f * float(n_tests);

    hipEventDestroy( start );
    hipEventDestroy( stop );

    fprintf(stderr, "lbvh bbox reduction test... done\n");
    fprintf(stderr, "  time       : %f ms\n", time * 1000.0f );
    fprintf(stderr, "  points/sec : %f M\n", (n_points / time) / 1.0e6f );
}

} // namespace sandbox

